// File: cuda_functions.cu

#include <hip/hip_runtime.h>

extern "C" {
    __global__ void add_arrays(float* a, float* b, float* c, int N) {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        if (index < N) {
            c[index] = a[index] + b[index];
        }
    }

    void call_cuda_function(float* a, float* b, float* c, int N) {
        float *d_a, *d_b, *d_c;

        hipMalloc(&d_a, N * sizeof(float));
        hipMalloc(&d_b, N * sizeof(float));
        hipMalloc(&d_c, N * sizeof(float));

        hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);

        int blockSize = 256;
        int numBlocks = (N + blockSize - 1) / blockSize;
        add_arrays<<<numBlocks, blockSize>>>(d_a, d_b, d_c, N);

        hipMemcpy(c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }
}

